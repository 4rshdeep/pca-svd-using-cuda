#include "lab3_io.h"
#include "lab3_cuda.h"

#include <stdlib.h>
#include <omp.h>

/*
	Arguments:
		arg1: input filename (consist M, N and D)
		arg2: retention (percentage of information to be retained by PCA)
*/


// double get_array_value(double* matrix, int row, int column, int num_col) {
//     // cout << column + row*num_col<< endl;
//     return matrix[ column+row*num_col ];
// }


int main(int argc, char const *argv[])
{
	if (argc < 3){
		printf("\nLess Arguments\n");
		return 0;
	}

	if (argc > 3){
		printf("\nTOO many Arguments\n");
		return 0;
	}

	//---------------------------------------------------------------------
	int M;			//no of rows (samples) in input matrix D (input)
	int N;			//no of columns (features) in input matrix D (input)
	double* D;		//1D array of M x N matrix to be reduced (input)
	double* U;		//1D array of N x N matrix U (to be computed by SVD)
	double* SIGMA;	//1D array of N x M diagonal matrix SIGMA (to be computed by SVD)
	double* V_T;		//1D array of M x M matrix V_T (to be computed by SVD)
	int K;			//no of coulmns (features) in reduced matrix D_HAT (to be computed by PCA)
	double *D_HAT;	//1D array of M x K reduced matrix (to be computed by PCA)
	int retention;	//percentage of information to be retained by PCA (command line input)
	//---------------------------------------------------------------------

	retention = atoi(argv[2]);	//retention = 90 means 90% of information should be retained

	float computation_time;

	/*
		-- Pre-defined function --
		reads matrix and its dimentions from input file and creats array D
	    #elements in D is M * N
        format - 
        --------------------------------------------------------------------------------------
        | D[0][0] | D[0][1] | ... | D[0][N-1] | D[1][0] | ... | D[1][N-1] | ... | D[M-1][N-1] |
        --------------------------------------------------------------------------------------
	*/
	read_matrix (argv[1], &M, &N, &D);

	// printf("M, N: %d %d\n", M, N);


	// exit(0);

	U = (double*) malloc(sizeof(double) * N*N);
	SIGMA = (double*) malloc(sizeof(double) * N);
	V_T = (double*) malloc(sizeof(double) * M*M);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	
	// /*
	// 	*****************************************************
	// 		TODO -- You must implement this function
	// 	*****************************************************
	// */
	SVD_and_PCA(M, N, D, &U, &SIGMA, &V_T, &D_HAT, &K, retention);
	bool to_print = true;
	if(to_print==true){
		double *temp = (double*)malloc(sizeof(double)*N*M);
		double *ans = (double*)malloc(sizeof(double)*N*M);
		for(int i=0;i<N;i++){
			for(int j=0;j<M;j++){
				temp[i*M+j] = U[i*N + j]*SIGMA[j];
				// for(int k=0;k<N;k++){
					// if(k==j){
						// temp[i*M+j] += U[i*N + k]*SIGMA[k];	
					// }
					// temp[i*M+j] += U[i*N + k]*SIGMA[k*M+j];
				// }
			}
		}
		printf("%s\n", "D_T is here");
		for(int i=0;i<N;i++){
			for(int j=0;j<M;j++){
				ans[i*M+j] = 0;
				for(int k=0;k<M;k++){
					ans[i*M+j] += temp[i*M + k]*V_T[k*M+j];
				}
				printf("%f ", ans[i*M+j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("%s %d\n", "K is ", K);
		printf("%s\n", "D_Hat");
		for(int i=0;i<M;i++){
			for(int j=0;j<K;j++){
				printf("%f ", D_HAT[i*K+j]);
			}
			printf("\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computation_time, start, stop);
	
	/*
		--Pre-defined functions --
		checks for correctness of results computed by SVD and PCA
		and outputs the results
	*/
	write_result(M, N, D, U, SIGMA, V_T, K, D_HAT, computation_time);


	// for (int i = 0; i < M; ++i)
	// {
	// 	for (int j = 0; j < K; ++j)
	// 	{
	// 		printf("%.2f ",  D_HAT[j+i*K]);
	// 	}
	// 	printf("\n");
	// }

	return 0;
}
